#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "function_cuda.h"

/* *** GPU Functions *** */

__device__ double func1(double *t_x,double *t_y,double *o){
  double retval;

  
  return retval;
}

__global__ void func_test(double *in,double *out){
  int tid = blockIdx.x;
  out[tid] = in[tid]*in[tid];
  return;
}



/* *** CPU Functions *** */

double *FunctionCuda::test(double *in,int N){
  
  double *ret = new double [N];
  double *dev_in, *dev_out;

  hipMalloc((void **)&dev_in,N*sizeof(double));
  hipMalloc((void **)&dev_out,N*sizeof(double));

  hipMemcpy(dev_in,in,N*sizeof(double),hipMemcpyHostToDevice);

  func_test<<<N,1>>>(dev_in,dev_out);
  
  hipMemcpy(ret,dev_out,N*sizeof(double),hipMemcpyDeviceToHost);
  
  /*
  for(int i=0;i<N;i++){
    ret[i] = in[i]*in[i];
  }
  */

  
  return ret;
}

