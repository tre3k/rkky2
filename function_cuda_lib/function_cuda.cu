#include "hip/hip_runtime.h"
/*
 * Kirill Pshenichny aka tre3k (c) 11.2018
 * This is part for calculationt the neutron scattering in the samples
 * with RKKY interaction.
 *
 */


#include <hip/hip_runtime.h>

#include "function_cuda.h"

/* *** GPU Functions *** */

__device__ double dE_i,dDeltaH,dA0,dk_s,dk_i;

__device__ double *dMap;                      // variable for dispersion 


struct sFunc{
  double func1,func2;
};

__global__ void calcConst(){
  dDeltaH = dDeltaH*P_g*P_uB;
  dE_i = P_h*P_h*dk_i*dk_i/2/P_mn;
  return;
}

__device__ struct sFunc func(double *lt_x,double *lt_y,double *lo){
  struct sFunc retval;

  double o = *lo;
  double t_x = *lt_x;
  double t_y = *lt_y;

  double o2 = o*o;
  double t_x2 = t_x*t_x;
  double t_y2 = t_y*t_y;

  double A = t_x2*dk_i*dk_i;
  double B = t_y2*dk_i*dk_i;
  double C = P_h*P_h*o2*dk_i*dk_i/4/dE_i/dE_i;
  double D = dk_s*dk_s; 

  retval.func1 = o - dA0*(A+B+C-D)*(A+B+C-D) - dDeltaH;
  retval.func2 = o + dA0*(A+B+C-D)*(A+B+C-D) + dDeltaH;
  
  return retval;
}

__global__ void generateMap(double *from_var, double *to_var, int *N){
  dMap = new double [*N];
  double dvar = (*to_var - *from_var)/ (double)(*N);
  for(int i=0;i<*N;i++) dMap[i] = *from_var + (double)i*dvar;
}

__global__ void deleteMap(){
  delete [] dMap;
}

__global__ void dispersionAtTheta_x(double *out){
  
  return;
}

__global__ void func_test(double *in,double *out){
  int tid = blockIdx.x;
  out[tid] = in[tid]*in[tid];

  return;
}



/* *** CPU Functions *** */

void FunctionCuda::memcpyConst(){

  double *lA0,*lDeltaH,*lk_i,*lk_s;
  lA0 = &dA0;
  lDeltaH = &dDeltaH;
  lk_i = &dk_i;
  lk_s = &dk_s;

  hipMemcpy(lA0, &A, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(lDeltaH, &DeltaH, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(lk_i, &k_i, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(lk_s, &k_s, sizeof(double), hipMemcpyHostToDevice);
  
  calcConst<<<1,1>>>();
  
  return;
}

void FunctionCuda::getDispersion(double *data1,double *data2,
				 int flags, double from_var, double to_var, int N){

  memcpyConst();

  
  return;
}

double *FunctionCuda::test(double *in,int N){
  
  double *ret = new double [N];
  double *dev_in, *dev_out;

  hipMalloc((void **)&dev_in,N*sizeof(double));
  hipMalloc((void **)&dev_out,N*sizeof(double));

  hipMemcpy(dev_in,in,N*sizeof(double),hipMemcpyHostToDevice);

  func_test<<<N,1>>>(dev_in,dev_out);
  
  hipMemcpy(ret,dev_out,N*sizeof(double),hipMemcpyDeviceToHost);
  
  /*
  for(int i=0;i<N;i++){
    ret[i] = in[i]*in[i];
  }
  */

  
  return ret;
}

